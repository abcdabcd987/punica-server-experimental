
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

static inline __device__ __host__ unsigned
__pack_half2(const half x, const half y) {
  unsigned v0 = *((unsigned short *)&x);
  unsigned v1 = *((unsigned short *)&y);
  return (v1 << 16) | v0;
}

extern "C" __global__ void __launch_bounds__(32) rotary_mha_decode_kvconst_12_64_12_2048_float16_kernel(half* __restrict__ K_proj, half* __restrict__ O, half* __restrict__ Q_proj, half* __restrict__ V_proj, half* __restrict__ kvbuf, int64_t* __restrict__ kvidx, int64_t* __restrict__ past_len, int64_t B, int64_t layer_idx, int64_t nnz) {
  __shared__ half Q_rotary[64];
  __shared__ half K_last_rotary[64];
  float m_now[1];
  float m_prev[1];
  float d_now[1];
  float d_prev[1];
  __shared__ float O_shared[64];
  __shared__ half K_shared[2048];
  float in_thread_X[1];
  float red_buf0[1];
  float red_buf0_1[1];
  __shared__ float A[32];
  float red_buf0_2[1];
  float o[1];
  #pragma unroll
  for (int64_t k_0 = 0; k_0 < 2L; ++k_0) {
    float emb = (((float)past_len[blockIdx.x]) / powf(1.000000e+04f, (((float)(threadIdx.x * 2L)) * 1.562500e-02f)));
    float cos = __cosf(emb);
    float sin = __sinf(emb);
    Q_rotary[((k_0 * 32L) + threadIdx.x)] = ((half)((((float)Q_proj[((((blockIdx.x * 768L) + (blockIdx.y * 64L)) + (k_0 * 32L)) + threadIdx.x)]) * cos) + (((float)((k_0 == 1L) ? Q_proj[(((((blockIdx.x * 768L) + (blockIdx.y * 64L)) + (k_0 * 32L)) + threadIdx.x) - 32L)] : (Q_proj[(((((blockIdx.x * 768L) + (blockIdx.y * 64L)) + (k_0 * 32L)) + threadIdx.x) + 32L)] * __float2half_rn(-1.000000e+00f)))) * sin)));
    K_last_rotary[((k_0 * 32L) + threadIdx.x)] = ((half)((((float)K_proj[((((blockIdx.x * 768L) + (blockIdx.y * 64L)) + (k_0 * 32L)) + threadIdx.x)]) * cos) + (((float)((k_0 == 1L) ? K_proj[(((((blockIdx.x * 768L) + (blockIdx.y * 64L)) + (k_0 * 32L)) + threadIdx.x) - 32L)] : (K_proj[(((((blockIdx.x * 768L) + (blockIdx.y * 64L)) + (k_0 * 32L)) + threadIdx.x) + 32L)] * __float2half_rn(-1.000000e+00f)))) * sin)));
  }
  __syncthreads();
  for (int64_t k_2_s = 0; k_2_s < 4L; ++k_2_s) {
    if (threadIdx.x < 16L) {
      kvbuf[((((((kvidx[blockIdx.x] * 37748736L) + (layer_idx * 3145728L)) + (past_len[blockIdx.x] * 768L)) + (blockIdx.y * 64L)) + (threadIdx.x * 4L)) + k_2_s)] = K_last_rotary[((threadIdx.x * 4L) + k_2_s)];
    }
  }
  for (int64_t k_2_s_1 = 0; k_2_s_1 < 4L; ++k_2_s_1) {
    if (threadIdx.x < 16L) {
      kvbuf[(((((((kvidx[blockIdx.x] * 37748736L) + (layer_idx * 3145728L)) + (past_len[blockIdx.x] * 768L)) + (blockIdx.y * 64L)) + (threadIdx.x * 4L)) + k_2_s_1) + 1572864L)] = V_proj[((((blockIdx.x * 768L) + (blockIdx.y * 64L)) + (threadIdx.x * 4L)) + k_2_s_1)];
    }
  }
  m_now[0] = -3.402823e+38f;
  m_prev[0] = -3.402823e+38f;
  d_now[0] = 0.000000e+00f;
  d_prev[0] = 0.000000e+00f;
  #pragma unroll
  for (int k_0_1 = 0; k_0_1 < 2; ++k_0_1) {
    O_shared[((((int64_t)k_0_1) * 32L) + threadIdx.x)] = 0.000000e+00f;
  }
  for (int64_t jo = 0; jo < ((past_len[blockIdx.x] + 31L) >> 5L); ++jo) {
    __syncthreads();
    #pragma unroll
    for (int64_t ji_k_fused_0 = 0; ji_k_fused_0 < 16L; ++ji_k_fused_0) {
      *(uint2*)(K_shared + ((ji_k_fused_0 * 128L) + (threadIdx.x * 4L))) = (((((jo * 32L) + (ji_k_fused_0 * 2L)) + (threadIdx.x >> 4L)) <= past_len[blockIdx.x]) ? *(uint2*)(kvbuf + (((((((kvidx[blockIdx.x] * 37748736L) + (layer_idx * 3145728L)) + (jo * 24576L)) + (ji_k_fused_0 * 1536L)) + ((threadIdx.x >> 4L) * 768L)) + (blockIdx.y * 64L)) + ((threadIdx.x & 15L) * 4L))) : make_uint2(__pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)), __pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f))));
    }
    __syncthreads();
    #pragma unroll
    for (int64_t ji = 0; ji < 32L; ++ji) {
      in_thread_X[0] = 0.000000e+00f;
      #pragma unroll
      for (int k_0_2 = 0; k_0_2 < 2; ++k_0_2) {
        in_thread_X[0] = (in_thread_X[0] + (((((float)Q_rotary[((((int64_t)k_0_2) * 32L) + threadIdx.x)]) * ((float)K_shared[(((ji * 64L) + (((int64_t)k_0_2) * 32L)) + threadIdx.x)])) * 1.250000e-01f) + ((past_len[blockIdx.x] < ((jo * 32L) + ji)) ? -3.402823e+38f : 0.000000e+00f)));
      }
      uint mask[1];
      float t0[1];
      red_buf0[0] = in_thread_X[0];
      mask[0] = __activemask();
      t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 16, 32);
      red_buf0[0] = (red_buf0[0] + t0[0]);
      t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 8, 32);
      red_buf0[0] = (red_buf0[0] + t0[0]);
      t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 4, 32);
      red_buf0[0] = (red_buf0[0] + t0[0]);
      t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
      red_buf0[0] = (red_buf0[0] + t0[0]);
      t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
      red_buf0[0] = (red_buf0[0] + t0[0]);
      red_buf0[0] = __shfl_sync(mask[0], red_buf0[0], 0, 32);
      if (threadIdx.x == 0L) {
        ((float*)K_last_rotary)[ji] = red_buf0[0];
      }
    }
    __syncthreads();
    #pragma unroll
    for (int64_t ji_k_fused_0_1 = 0; ji_k_fused_0_1 < 16L; ++ji_k_fused_0_1) {
      *(uint2*)(K_shared + ((ji_k_fused_0_1 * 128L) + (threadIdx.x * 4L))) = (((((jo * 32L) + (ji_k_fused_0_1 * 2L)) + (threadIdx.x >> 4L)) <= past_len[blockIdx.x]) ? *(uint2*)(kvbuf + ((((((((kvidx[blockIdx.x] * 37748736L) + (layer_idx * 3145728L)) + (jo * 24576L)) + (ji_k_fused_0_1 * 1536L)) + ((threadIdx.x >> 4L) * 768L)) + (blockIdx.y * 64L)) + ((threadIdx.x & 15L) * 4L)) + 1572864L)) : make_uint2(__pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)), __pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f))));
    }
    uint mask_1[1];
    float t0_1[1];
    red_buf0_1[0] = ((float*)K_last_rotary)[threadIdx.x];
    mask_1[0] = __activemask();
    t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 16, 32);
    red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
    t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 8, 32);
    red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
    t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 4, 32);
    red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
    t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 2, 32);
    red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
    t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 1, 32);
    red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
    red_buf0_1[0] = __shfl_sync(mask_1[0], red_buf0_1[0], 0, 32);
    m_prev[0] = m_now[0];
    m_now[0] = max(m_prev[0], red_buf0_1[0]);
    A[threadIdx.x] = __expf((((float*)K_last_rotary)[threadIdx.x] - m_now[0]));
    uint mask_2[1];
    float t0_2[1];
    red_buf0_2[0] = A[threadIdx.x];
    mask_2[0] = __activemask();
    t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 16, 32);
    red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
    t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 8, 32);
    red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
    t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 4, 32);
    red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
    t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 2, 32);
    red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
    t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 1, 32);
    red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
    red_buf0_2[0] = __shfl_sync(mask_2[0], red_buf0_2[0], 0, 32);
    d_prev[0] = d_now[0];
    d_now[0] = ((d_prev[0] * __expf((m_prev[0] - m_now[0]))) + max(red_buf0_2[0], 1.000000e-05f));
    __syncthreads();
    #pragma unroll
    for (int k_0_3 = 0; k_0_3 < 2; ++k_0_3) {
      #pragma unroll
      for (int ji_1 = 0; ji_1 < 32; ++ji_1) {
        if (ji_1 == 0) {
          o[0] = 0.000000e+00f;
        }
        o[0] = (o[0] + (A[ji_1] * ((float)K_shared[(((((int64_t)ji_1) * 64L) + (((int64_t)k_0_3) * 32L)) + threadIdx.x)])));
      }
      O_shared[((((int64_t)k_0_3) * 32L) + threadIdx.x)] = ((((O_shared[((((int64_t)k_0_3) * 32L) + threadIdx.x)] * d_prev[0]) * __expf((m_prev[0] - m_now[0]))) + o[0]) / d_now[0]);
    }
  }
  __syncthreads();
  for (int64_t k_2_s_2 = 0; k_2_s_2 < 4L; ++k_2_s_2) {
    if (threadIdx.x < 16L) {
      O[((((blockIdx.x * 768L) + (blockIdx.y * 64L)) + (threadIdx.x * 4L)) + k_2_s_2)] = ((half)O_shared[((threadIdx.x * 4L) + k_2_s_2)]);
    }
  }
}

extern "C" void launch_rotary_mha_decode_kvconst_12_64_12_2048_float16_kernel(void* __restrict__ K_proj, void* __restrict__ O, void* __restrict__ Q_proj, void* __restrict__ V_proj, void* __restrict__ kvbuf, void* __restrict__ kvidx, void* __restrict__ past_len, int64_t B, int64_t layer_idx, int64_t nnz) {
  dim3 grid(B, 12);
  dim3 block(32);
  rotary_mha_decode_kvconst_12_64_12_2048_float16_kernel<<<grid, block>>>((half*)K_proj, (half*)O, (half*)Q_proj, (half*)V_proj, (half*)kvbuf, (int64_t*)kvidx, (int64_t*)past_len, B, layer_idx, nnz);
}