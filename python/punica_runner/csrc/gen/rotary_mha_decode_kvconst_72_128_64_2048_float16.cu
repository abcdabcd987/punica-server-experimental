
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

static inline __device__ __host__ unsigned
__pack_half2(const half x, const half y) {
  unsigned v0 = *((unsigned short *)&x);
  unsigned v1 = *((unsigned short *)&y);
  return (v1 << 16) | v0;
}

extern "C" __global__ void __launch_bounds__(32) rotary_mha_decode_kvconst_72_128_64_2048_float16_kernel(half* __restrict__ K_proj, half* __restrict__ O, half* __restrict__ Q_proj, half* __restrict__ V_proj, half* __restrict__ kvbuf, int64_t* __restrict__ kvidx, int64_t* __restrict__ past_len, int64_t B, int64_t layer_idx, int64_t nnz) {
  __shared__ half Q_rotary[128];
  __shared__ half K_last_rotary[128];
  float m_now[1];
  float m_prev[1];
  float d_now[1];
  float d_prev[1];
  __shared__ float O_shared[128];
  __shared__ half K_shared[4096];
  float in_thread_X[1];
  float red_buf0[1];
  float red_buf0_1[1];
  __shared__ float A[32];
  float red_buf0_2[1];
  float o[1];
  #pragma unroll
  for (int64_t k_0 = 0; k_0 < 4L; ++k_0) {
    float emb = (((float)past_len[blockIdx.x]) / powf(1.000000e+04f, (((float)(((k_0 & 1L) * 64L) + (threadIdx.x * 2L))) * 7.812500e-03f)));
    float cos = __cosf(emb);
    float sin = __sinf(emb);
    Q_rotary[((k_0 * 32L) + threadIdx.x)] = ((half)((((float)Q_proj[((((blockIdx.x * 9216L) + (blockIdx.y * 128L)) + (k_0 * 32L)) + threadIdx.x)]) * cos) + (((float)((2L <= k_0) ? Q_proj[(((((blockIdx.x * 9216L) + (blockIdx.y * 128L)) + (k_0 * 32L)) + threadIdx.x) - 64L)] : (Q_proj[(((((blockIdx.x * 9216L) + (blockIdx.y * 128L)) + (k_0 * 32L)) + threadIdx.x) + 64L)] * __float2half_rn(-1.000000e+00f)))) * sin)));
    K_last_rotary[((k_0 * 32L) + threadIdx.x)] = ((half)((((float)K_proj[((((blockIdx.x * 9216L) + (blockIdx.y * 128L)) + (k_0 * 32L)) + threadIdx.x)]) * cos) + (((float)((2L <= k_0) ? K_proj[(((((blockIdx.x * 9216L) + (blockIdx.y * 128L)) + (k_0 * 32L)) + threadIdx.x) - 64L)] : (K_proj[(((((blockIdx.x * 9216L) + (blockIdx.y * 128L)) + (k_0 * 32L)) + threadIdx.x) + 64L)] * __float2half_rn(-1.000000e+00f)))) * sin)));
  }
  __syncthreads();
  *(uint2*)(kvbuf + (((((kvidx[blockIdx.x] * 2415919104L) + (layer_idx * 37748736L)) + (past_len[blockIdx.x] * 9216L)) + (blockIdx.y * 128L)) + (threadIdx.x * 4L))) = *(uint2*)(K_last_rotary + (threadIdx.x * 4L));
  *(uint2*)(kvbuf + ((((((kvidx[blockIdx.x] * 2415919104L) + (layer_idx * 37748736L)) + (past_len[blockIdx.x] * 9216L)) + (blockIdx.y * 128L)) + (threadIdx.x * 4L)) + 18874368L)) = *(uint2*)(V_proj + (((blockIdx.x * 9216L) + (blockIdx.y * 128L)) + (threadIdx.x * 4L)));
  m_now[0] = -3.402823e+38f;
  m_prev[0] = -3.402823e+38f;
  d_now[0] = 0.000000e+00f;
  d_prev[0] = 0.000000e+00f;
  #pragma unroll
  for (int k_0_1 = 0; k_0_1 < 4; ++k_0_1) {
    O_shared[((((int64_t)k_0_1) * 32L) + threadIdx.x)] = 0.000000e+00f;
  }
  for (int64_t jo = 0; jo < ((past_len[blockIdx.x] + 31L) >> 5L); ++jo) {
    __syncthreads();
    #pragma unroll
    for (int64_t ji_k_fused_0 = 0; ji_k_fused_0 < 32L; ++ji_k_fused_0) {
      *(uint2*)(K_shared + ((ji_k_fused_0 * 128L) + (threadIdx.x * 4L))) = ((((jo * 32L) + ji_k_fused_0) <= past_len[blockIdx.x]) ? *(uint2*)(kvbuf + ((((((kvidx[blockIdx.x] * 2415919104L) + (layer_idx * 37748736L)) + (jo * 294912L)) + (ji_k_fused_0 * 9216L)) + (blockIdx.y * 128L)) + (threadIdx.x * 4L))) : make_uint2(__pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)), __pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f))));
    }
    __syncthreads();
    #pragma unroll
    for (int64_t ji = 0; ji < 32L; ++ji) {
      in_thread_X[0] = 0.000000e+00f;
      #pragma unroll
      for (int k_0_2 = 0; k_0_2 < 4; ++k_0_2) {
        in_thread_X[0] = (in_thread_X[0] + (((((float)Q_rotary[((((int64_t)k_0_2) * 32L) + threadIdx.x)]) * ((float)K_shared[(((ji * 128L) + (((int64_t)k_0_2) * 32L)) + threadIdx.x)])) * 8.838835e-02f) + ((past_len[blockIdx.x] < ((jo * 32L) + ji)) ? -3.402823e+38f : 0.000000e+00f)));
      }
      uint mask[1];
      float t0[1];
      red_buf0[0] = in_thread_X[0];
      mask[0] = __activemask();
      t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 16, 32);
      red_buf0[0] = (red_buf0[0] + t0[0]);
      t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 8, 32);
      red_buf0[0] = (red_buf0[0] + t0[0]);
      t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 4, 32);
      red_buf0[0] = (red_buf0[0] + t0[0]);
      t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
      red_buf0[0] = (red_buf0[0] + t0[0]);
      t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
      red_buf0[0] = (red_buf0[0] + t0[0]);
      red_buf0[0] = __shfl_sync(mask[0], red_buf0[0], 0, 32);
      if (threadIdx.x == 0L) {
        ((float*)K_last_rotary)[ji] = red_buf0[0];
      }
    }
    __syncthreads();
    #pragma unroll
    for (int64_t ji_k_fused_0_1 = 0; ji_k_fused_0_1 < 32L; ++ji_k_fused_0_1) {
      *(uint2*)(K_shared + ((ji_k_fused_0_1 * 128L) + (threadIdx.x * 4L))) = ((((jo * 32L) + ji_k_fused_0_1) <= past_len[blockIdx.x]) ? *(uint2*)(kvbuf + (((((((kvidx[blockIdx.x] * 2415919104L) + (layer_idx * 37748736L)) + (jo * 294912L)) + (ji_k_fused_0_1 * 9216L)) + (blockIdx.y * 128L)) + (threadIdx.x * 4L)) + 18874368L)) : make_uint2(__pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)), __pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f))));
    }
    uint mask_1[1];
    float t0_1[1];
    red_buf0_1[0] = ((float*)K_last_rotary)[threadIdx.x];
    mask_1[0] = __activemask();
    t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 16, 32);
    red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
    t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 8, 32);
    red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
    t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 4, 32);
    red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
    t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 2, 32);
    red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
    t0_1[0] = __shfl_down_sync(mask_1[0], red_buf0_1[0], 1, 32);
    red_buf0_1[0] = max(red_buf0_1[0], t0_1[0]);
    red_buf0_1[0] = __shfl_sync(mask_1[0], red_buf0_1[0], 0, 32);
    m_prev[0] = m_now[0];
    m_now[0] = max(m_prev[0], red_buf0_1[0]);
    A[threadIdx.x] = __expf((((float*)K_last_rotary)[threadIdx.x] - m_now[0]));
    uint mask_2[1];
    float t0_2[1];
    red_buf0_2[0] = A[threadIdx.x];
    mask_2[0] = __activemask();
    t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 16, 32);
    red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
    t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 8, 32);
    red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
    t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 4, 32);
    red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
    t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 2, 32);
    red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
    t0_2[0] = __shfl_down_sync(mask_2[0], red_buf0_2[0], 1, 32);
    red_buf0_2[0] = (red_buf0_2[0] + t0_2[0]);
    red_buf0_2[0] = __shfl_sync(mask_2[0], red_buf0_2[0], 0, 32);
    d_prev[0] = d_now[0];
    d_now[0] = ((d_prev[0] * __expf((m_prev[0] - m_now[0]))) + max(red_buf0_2[0], 1.000000e-05f));
    __syncthreads();
    #pragma unroll
    for (int k_0_3 = 0; k_0_3 < 4; ++k_0_3) {
      #pragma unroll
      for (int ji_1 = 0; ji_1 < 32; ++ji_1) {
        if (ji_1 == 0) {
          o[0] = 0.000000e+00f;
        }
        o[0] = (o[0] + (A[ji_1] * ((float)K_shared[(((((int64_t)ji_1) * 128L) + (((int64_t)k_0_3) * 32L)) + threadIdx.x)])));
      }
      O_shared[((((int64_t)k_0_3) * 32L) + threadIdx.x)] = ((((O_shared[((((int64_t)k_0_3) * 32L) + threadIdx.x)] * d_prev[0]) * __expf((m_prev[0] - m_now[0]))) + o[0]) / d_now[0]);
    }
  }
  __syncthreads();
  uint2 __1;
  float4 v_ = *(float4*)(O_shared + (threadIdx.x * 4L));
  ((half2*)(&(__1.x)))->x = (half)(v_.x);
  ((half2*)(&(__1.x)))->y = (half)(v_.y);
  ((half2*)(&(__1.y)))->x = (half)(v_.z);
  ((half2*)(&(__1.y)))->y = (half)(v_.w);
  *(uint2*)(O + (((blockIdx.x * 9216L) + (blockIdx.y * 128L)) + (threadIdx.x * 4L))) = __1;
}

extern "C" void launch_rotary_mha_decode_kvconst_72_128_64_2048_float16_kernel(void* __restrict__ K_proj, void* __restrict__ O, void* __restrict__ Q_proj, void* __restrict__ V_proj, void* __restrict__ kvbuf, void* __restrict__ kvidx, void* __restrict__ past_len, int64_t B, int64_t layer_idx, int64_t nnz) {
  dim3 grid(B, 72);
  dim3 block(32);
  rotary_mha_decode_kvconst_72_128_64_2048_float16_kernel<<<grid, block>>>((half*)K_proj, (half*)O, (half*)Q_proj, (half*)V_proj, (half*)kvbuf, (int64_t*)kvidx, (int64_t*)past_len, B, layer_idx, nnz);
}